
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <cstdlib>
#include <math.h>
#include <chrono>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>

#define gNPOINTS 100000000
#define cNPOINTS 100000000

__global__
void gpuMontyCarlo(int *gpuInCricle, double *randomNums)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double xPoint = randomNums[idx];
    double yPoint = randomNums[idx+gNPOINTS];

    double radius = (xPoint*xPoint) + (yPoint*yPoint);
    if(radius <= 1)
    {
        gpuInCricle[idx] = 1;
    }
    else
    {
        gpuInCricle[idx] = 0;
    }
};

void cpuMontyCarlo(double &cpuPI)
{
    std::mt19937 rng(time(NULL));
    std::uniform_real_distribution<double> randomPoint(0.0,1.0);

    int inCircle = 0;
    double radius;
    double xPoint;
    double yPoint;
    
    for(int x = 0; x < cNPOINTS; x++)
    {
        xPoint = randomPoint(rng);
        yPoint = randomPoint(rng);
        //radius of unit circle is x^2+y^2 = 1
        radius = (xPoint*xPoint) + (yPoint*yPoint);

        if(radius <= 1)
        {
            inCircle++;
        }
    }
    cpuPI = 4.0*((double)inCircle/cNPOINTS);
};

int main()
{
    // CPU ---------------------------------------------------------------------------------------------------------------
    double cpuPI;
    auto cpuStart = std::chrono::system_clock::now();

    cpuMontyCarlo(cpuPI);

    auto cpuEnd = std::chrono::system_clock::now();

    // GPU ---------------------------------------------------------------------------------------------------------------
    
    //Setup gpu random number states
    double gpuPI;
    int *gpuInCricle;
    double *randomNums;
    hipMallocManaged(&gpuInCricle, gNPOINTS*sizeof(int));
    hipMallocManaged(&randomNums, gNPOINTS*2*sizeof(double));

    std::mt19937 rng(time(NULL));
    std::uniform_real_distribution<double> randomPoint(0.0,1.0);
    for(int x = 0; x < gNPOINTS*2; x++)
    {
        randomNums[x] = randomPoint(rng);
    }

    auto gpuStart = std::chrono::system_clock::now();

    gpuMontyCarlo<<<gNPOINTS/1000, 1000>>>(gpuInCricle, randomNums);
    hipDeviceSynchronize();

    auto gpuEnd = std::chrono::system_clock::now();

    int gInCricle = 0;
    for(int x = 0; x < gNPOINTS; x++)
    {
        if(gpuInCricle[x] == 1)
        {
            gInCricle++;
        }
    }
    gpuPI = 4.0*((double)gInCricle/gNPOINTS);

    std::chrono::duration<double> cpuTime = cpuEnd - cpuStart;
    std::chrono::duration<double> gpuTime = gpuEnd - gpuStart;
    std::cout << "CPU PI: " << cpuPI << " : TIME " << cpuTime.count() << std::endl;
    std::cout << "GPU PI: " << gpuPI << " : TIME " << gpuTime.count() << std::endl;

    hipFree(gpuInCricle);
    hipFree(randomNums);
    return 0;
}